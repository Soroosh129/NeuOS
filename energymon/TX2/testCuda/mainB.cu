#include "hip/hip_runtime.h"
// System Includes
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <energymon/energymon-odroid.h>
//#include <energymon/energymon-default.h>
// Cuda Includes
//#include <hip/hip_runtime.h>
//#include <hipblas.h>
//#include <hipblas.h>
//#include <hip/hip_runtime.h>

// auxiliary functions
#include "AuxFuncs.h"

// Macro to store elements in a linear space in row-major format
#define IDX2C(i, j, ld) (((i) * (ld)) + (j))
#define BLOCK_SIZE 512

__global__ void MultMVNaiveKernel(double *A, double *b, double *c, const int M, const int N){
	int row = threadIdx.x + blockIdx.x * blockDim.x;

	double sum = 0;
	if(row < M) {
		for(int i = 0; i < N; i++) {
			sum += b[i] * A[row * N + i];
		}
		c[row] = sum;
	}
}


int do_work(int argc, char ** argv) {
	int M, N;
	// init the seed with current local time
	srand(time(NULL));

	// Get M - N values from arguments
	if (argc == 3) {
		M = atoi(argv[1]);
		N = atoi(argv[2]);
	}
	else {
		fprintf(stderr, "Insufficient command line arguments!\n");
		fprintf(stderr, "USAGE: main <matrixHeight> <matrixWidth>\n");
		exit(-1);
	}

	double * h_A, * h_b, * h_c; // host copies of a, b, c
	double * d_A, * d_b, * d_c; // device copies of a, b, c
	d_A = d_b = d_c = 0;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Allocate host memory for the matrix and the vectors
	((h_A = (double *) malloc(M * N * sizeof(double))) != 0) ?
	((h_b = (double *) malloc(N * sizeof(double))) != 0) ?
	((h_c = (double *) malloc(M * sizeof(double))) != 0) ?
	:
	_error_handler("host memory allocation error (C)\n") :
	_error_handler("host memory allocation error (B)\n") :
	_error_handler("host memory allocation error (A)\n") ;

	// Allocate device memory for the matrix and the vectors
	hipMalloc((void **) &d_A, sizeof(double) * M * N);
	hipMalloc((void **) &d_b, sizeof(double) * N);
	hipMalloc((void **) &d_c, sizeof(double) * M);
	
	// Initialize matrix A and vector b with some values and also zero-ize c vector
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			h_A[i*N + j] = randDouble();
		}
	}

	for (int i = 0; i < N; i++) {
		h_b[i] = randDouble();
	}

	for (int i = 0; i < M; i++) {
		h_c[i] = 0;
	}

	hipEventRecord(start);
	
	// Copy data from host to device
	hipMemcpy(d_A, h_A, sizeof(double) * M * N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(double) * N, hipMemcpyHostToDevice);
	//hipMemcpy(d_c, h_c, sizeof(double) * M, hipMemcpyHostToDevice);

	unsigned int numOfBlocks = M / BLOCK_SIZE + 1;	   

	// Run kernel and measure the time needed
	//MultMVNaiveKernel<<<numOfBlocks, BLOCK_SIZE>>>(d_A, d_b, d_c, M, N);
	// Get results from the device
	hipMemcpy(h_c, d_c, M * sizeof(h_c[0]), hipMemcpyDeviceToHost);
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	fprintf(stdout, "Execution completed. Elapsed Time = %6.8f msecs\n", milliseconds);



	// Free host memory
	free(h_A); free(h_b); free(h_c);
	// Free GPU memory
	hipFree(d_A); hipFree(d_b); hipFree(d_c);

	return EXIT_SUCCESS;
}

int main(int argc, char** argv){
	energymon em;
	uint64_t start_nj[4], end_nj[4];
	for(int i =0;i<4;i++){
		start_nj[i]=0;
		end_nj[i]=0;
	}

	// get the energymon instance and initialize
	//energymon_get_default(&em);
	energymon_get_odroid(&em);
	em.finit(&em);

	// profile application function
	uint64_t start = em.fread(&em, start_nj);
	
	do_work(argc, argv);
	uint64_t end = em.fread(&em, end_nj);
	printf("========= %lld\n", start);
	printf("========= %lld\n", end);

	printf("Total energy in mJ: %f\n", (end_nj[0]-start_nj[0])/1000000.0);
	printf("Total energy in mJ: %f\n", (end_nj[1]-start_nj[1])/1000000.0);
	printf("Total energy in mJ: %f\n", (end_nj[2]-start_nj[2])/1000000.0);
	printf("Total energy in mJ: %f\n", (end_nj[3]-start_nj[3])/1000000.0);

	// destroy the instance
	em.ffinish(&em);
	return 0;
}
