#include "hip/hip_runtime.h"
// System Includes
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

// Cuda Includes
//#include <hip/hip_runtime.h>
//#include <hipblas.h>
//#include <hipblas.h>
//#include <hip/hip_runtime.h>

// auxiliary functions
#include "AuxFuncs.h"

// Macro to store elements in a linear space in row-major format
#define IDX2C(i, j, ld) (((i) * (ld)) + (j))
#define BLOCK_SIZE 512

__global__ void MultMVNaiveKernel(double *A, double *b, double *c, const int M, const int N){
	int row = threadIdx.x + blockIdx.x * blockDim.x;

	double sum = 0;
	if(row < M) {
		for(int i = 0; i < N; i++) {
			sum += b[i] * A[row * N + i];
		}
		c[row] = sum;
	}
}


int main(int argc, char ** argv) {
	int M, N;
	// init the seed with current local time
	srand(time(NULL));

	// Get M - N values from arguments
	if (argc == 3) {
		M = atoi(argv[1]);
		N = atoi(argv[2]);
	}
	else {
		fprintf(stderr, "Insufficient command line arguments!\n");
		fprintf(stderr, "USAGE: main <matrixHeight> <matrixWidth>\n");
		exit(-1);
	}

	double * h_A, * h_b, * h_c; // host copies of a, b, c
	double * d_A, * d_b, * d_c; // device copies of a, b, c
	d_A = d_b = d_c = 0;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Allocate host memory for the matrix and the vectors
	((h_A = (double *) malloc(M * N * sizeof(double))) != 0) ?
	((h_b = (double *) malloc(N * sizeof(double))) != 0) ?
	((h_c = (double *) malloc(M * sizeof(double))) != 0) ?
	:
	_error_handler("host memory allocation error (C)\n") :
	_error_handler("host memory allocation error (B)\n") :
	_error_handler("host memory allocation error (A)\n") ;

	// Allocate device memory for the matrix and the vectors
	hipMalloc((void **) &d_A, sizeof(double) * M * N);
	hipMalloc((void **) &d_b, sizeof(double) * N);
	hipMalloc((void **) &d_c, sizeof(double) * M);
	
	// Initialize matrix A and vector b with some values and also zero-ize c vector
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			h_A[i*N + j] = randDouble();
		}
	}

	for (int i = 0; i < N; i++) {
		h_b[i] = randDouble();
	}

	for (int i = 0; i < M; i++) {
		h_c[i] = 0;
	}

	// Copy data from host to device
	hipMemcpy(d_A, h_A, sizeof(double) * M * N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(double) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_c, h_c, sizeof(double) * M, hipMemcpyHostToDevice);

	unsigned int numOfBlocks = M / BLOCK_SIZE + 1;	   

	// Run kernel and measure the time needed
	hipEventRecord(start);
	//MultMVNaiveKernel<<<numOfBlocks, BLOCK_SIZE>>>(d_A, d_b, d_c, M, N);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	fprintf(stdout, "Execution completed. Elapsed Time = %6.8f msecs\n", milliseconds);

	// Get results from the device
	hipMemcpy(h_c, d_c, M * sizeof(h_c[0]), hipMemcpyDeviceToHost);
	
	// Free host memory
	free(h_A); free(h_b); free(h_c);
	// Free GPU memory
	hipFree(d_A); hipFree(d_b); hipFree(d_c);

	return EXIT_SUCCESS;
}

