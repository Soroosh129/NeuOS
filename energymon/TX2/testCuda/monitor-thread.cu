#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <pthread.h>
#include <time.h>
#include <unistd.h>
#include <math.h>
#include <energymon/energymon-odroid.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>


using namespace std;

// auxiliary functions
#include "AuxFuncs.h"

// Macro to store elements in a linear space in row-major format
#define IDX2C(i, j, ld) (((i) * (ld)) + (j))
#define BLOCK_SIZE 512

__global__ void MultMVNaiveKernel(double *A, double *b, double *c, const int M, const int N){
	int row = threadIdx.x + blockIdx.x * blockDim.x;

	double sum = 0;
	if(row < M) {
		for(int i = 0; i < N; i++) {
			sum += b[i] * A[row * N + i];
		}
		c[row] = sum;
	}
}


void do_work(int iter,	float  interval) {// interval in ms
	printf("do_work\n");
	int M=2048, N=2048;
	// init the seed with current local time
	srand(time(NULL));

	// Get M - N values from arguments
	//M = atoi(argv[1]);
	//N = atoi(argv[2]);

	double * h_A, * h_b, * h_c; // host copies of a, b, c
	double * d_A, * d_b, * d_c; // device copies of a, b, c
	d_A = d_b = d_c = 0;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Allocate host memory for the matrix and the vectors
	((h_A = (double *) malloc(M * N * sizeof(double))) != 0) ?
	((h_b = (double *) malloc(N * sizeof(double))) != 0) ?
	((h_c = (double *) malloc(M * sizeof(double))) != 0) ?
	:
	_error_handler("host memory allocation error (C)\n") :
	_error_handler("host memory allocation error (B)\n") :
	_error_handler("host memory allocation error (A)\n") ;

	// Allocate device memory for the matrix and the vectors
	hipMalloc((void **) &d_A, sizeof(double) * M * N);
	hipMalloc((void **) &d_b, sizeof(double) * N);
	hipMalloc((void **) &d_c, sizeof(double) * M);
	
	// Initialize matrix A and vector b with some values and also zero-ize c vector
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			h_A[i*N + j] = randDouble();
		}
	}

	for (int i = 0; i < N; i++) {
		h_b[i] = randDouble();
	}

	for (int i = 0; i < M; i++) {
		h_c[i] = 0;
	}

	hipEventRecord(start);
	
	// Copy data from host to device
	hipMemcpy(d_A, h_A, sizeof(double) * M * N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(double) * N, hipMemcpyHostToDevice);
	//hipMemcpy(d_c, h_c, sizeof(double) * M, hipMemcpyHostToDevice);

	unsigned int numOfBlocks = M / BLOCK_SIZE + 1;	   

	// Run kernel and measure the time needed
	//
	for(int i=0;i<iter;i++){
		MultMVNaiveKernel<<<numOfBlocks, BLOCK_SIZE>>>(d_A, d_b, d_c, M, N);
		usleep(interval * 1000);
	}
	// Get results from the device
	hipMemcpy(h_c, d_c, M * sizeof(h_c[0]), hipMemcpyDeviceToHost);
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	fprintf(stdout, "Execution completed. Elapsed Time = %6.8f msecs\n", milliseconds);



	// Free host memory
	free(h_A); free(h_b); free(h_c);
	// Free GPU memory
	hipFree(d_A); hipFree(d_b); hipFree(d_c);
}

void* do_monitor(void* stime){
	float sleeptime = *((float*)stime)*1000000.0;
	printf("Start monitoring\n");
	energymon em;
	energymon_get_odroid(&em);
	em.finit(&em);

	uint64_t start_nj[4], end_nj[4];
	
	for(int i =0;i<4;i++){
		start_nj[i]=0;
		end_nj[i]=0;
	}

	usleep(sleeptime);	

	uint64_t end = em.fread(&em, end_nj);
	printf("#### %f %f %f %f\n", 
			(end_nj[0]-start_nj[0])/1000000.0, 
			(end_nj[1]-start_nj[1])/1000000.0, 
			(end_nj[2]-start_nj[2])/1000000.0, 
			(end_nj[3]-start_nj[3])/1000000.0);
	em.ffinish(&em);
	printf("End monitoring\n");
}
int main(int argc, char** argv){
	pthread_t monitor;
	float sleeptime =  atof(argv[1]); // in second
	int iter = atoi(argv[2]); 
	float interval = atof(argv[3]); // in millionsecond
	pthread_create(&monitor, NULL, do_monitor, (void*) &sleeptime);
	
	do_work(iter, interval);
	
	void* status;
	pthread_join(monitor,&status);
	return 0;
}
